#include "hip/hip_runtime.h"
#include <stdlib.h>

#include "linear_layer.hh"
#include "nn_exception.hh"
#include "nn_utils.hh"

__global__ void weightedSum(float* A, float* W, float* Z,
									 int A_x_dim, int A_y_dim,
									 int W_x_dim, int W_y_dim) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index < W_x_dim * W_y_dim) {
		int W_x = index % W_x_dim;
		int W_y = index / W_x_dim;

		int Z_x_dim = A_x_dim;
		int Z_y = W_y;
		int Z_x = 0;

		int A_y = W_x;
		float product_val = 0;

		for (int A_x = 0; A_x < A_x_dim; A_x++) {
			Z_x = A_x;
			product_val = W[index] * A[A_y * A_x_dim + A_x];
			atomicAdd(&Z[Z_y * Z_x_dim + Z_x], product_val);
		}
	}
}

__global__ void addBias(float* Z, float* b, int Z_x_dim, int Z_y_dim) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index < Z_x_dim * Z_y_dim) {
		int row = static_cast<int>(index / Z_x_dim);
		int col = index % Z_x_dim;
		Z[index] += b[row];
	}
}

LinearLayer::LinearLayer(std::string name, nn_utils::Shape W_shape) :
	W(W_shape), Z(), b(W_shape.y, 1)
{
	this->name = name;
	b.allocateCudaMemory();
	nn_utils::throwIfDeviceErrorsOccurred("Cannot initialize layer bias.");
	W.allocateCudaMemory();
	nn_utils::throwIfDeviceErrorsOccurred("Cannot initialize layer weights.");

	initializeBiasWithZeros();
	initializeWeightsRandomly();
}

void LinearLayer::initializeWeightsRandomly() {
	for (int x = 0; x < W.shape.x; x++) {
		for (int y = 0; y < W.shape.y; y++) {
			W.data[y * W.shape.x + x] = (static_cast<float>(rand()) / RAND_MAX) * weights_init_threshold;
		}
	}
}

void LinearLayer::initializeBiasWithZeros() {
	for (int x = 0; x < b.shape.x; x++) {
		b.data[x] = 0;
	}
}

LinearLayer::~LinearLayer() {
	W.freeCudaMemory();
	Z.freeCudaMemory();
}

nn_utils::Tensor3D LinearLayer::forward(nn_utils::Tensor3D A) {

	// TODO: should be initialized only once, not with every forward() call
	hipMallocManaged(&Z.data, W.shape.y * A.shape.x * sizeof(float));

	if (W.shape.x != A.shape.y) {
		throw NNException("Weight matrix and input matrix don't match.");
	}

	hipMemset(Z.data, 0, Z.shape.x * Z.shape.y * sizeof(float));
	Z.shape = nn_utils::Shape(A.shape.x, W.shape.y);

	dim3 block_size(256);
	dim3 num_of_blocks((W.shape.y * W.shape.x + block_size.x - 1) / block_size.x);
	weightedSum<<<block_size, num_of_blocks>>>(A.data, W.data, Z.data,
											   A.shape.x, A.shape.y,
											   W.shape.x, W.shape.y);
	hipDeviceSynchronize();
	nn_utils::throwIfDeviceErrorsOccurred("Cannot perform linear forward prop.");

	block_size.x = 256;
	num_of_blocks.x = ((Z.shape.y * Z.shape.x + block_size.x - 1) / block_size.x);
	addBias<<<block_size, num_of_blocks>>>(Z.data, b.data, Z.shape.x, Z.shape.y);
	hipDeviceSynchronize();
	nn_utils::throwIfDeviceErrorsOccurred("Cannot perform linear forward prop.");

	return Z;
}

int LinearLayer::getXDim() const {
	return W.shape.x;
}

int LinearLayer::getYDim() const {
	return W.shape.y;
}

const nn_utils::Tensor3D LinearLayer::getWeightsMatrix() const {
	return W;
}
