#include "hip/hip_runtime.h"
#include "nn_utils.hh"
#include "nn_exception.hh"

#include <math.h>
#include <iostream>

namespace nn_utils {

	void throwIfDeviceErrorsOccurred(const char* exception_message) {
		hipError_t error = hipGetLastError();
		if (error != hipSuccess) {
			throw NNException(exception_message);
		}
	}

	Shape::Shape(size_t x, size_t y, size_t z) :
		x(x), y(y), z(z)
	{ }

	Tensor3D::Tensor3D(size_t x_dim, size_t y_dim, size_t z_dim) :
		shape(x_dim, y_dim, z_dim), data(nullptr)
	{ }

	Tensor3D::Tensor3D(Shape shape) :
		shape(shape), data(nullptr)
	{ }

	void Tensor3D::allocateCudaMemory() {
		hipMallocManaged(&data, shape.x * shape.y * shape.z * sizeof(float));
		throwIfDeviceErrorsOccurred("Cannot allocate CUDA memory for Tensor3D.");
	}

	void Tensor3D::freeCudaMemory() {
		hipFree(data);
		data = nullptr;
	}

	float binaryCrossEntropyCost(nn_utils::Tensor3D predictions, nn_utils::Tensor3D target) {
		if (predictions.shape.x != target.shape.x) {
			throw NNException("Predictions and target shapes don't match.");
		}

		float cost = 0.0;
		for (int i = 0; i < predictions.shape.x; i++) {
			cost += target.data[i] * log(predictions.data[i]) + (1 - target.data[i]) * log(1 - predictions.data[i]);
		}

		return -cost / predictions.shape.x;
	}

	// TODO: move operation to CUDA
	Tensor3D dBinaryCrossEntropyCost(nn_utils::Tensor3D predictions, nn_utils::Tensor3D target) {
		if (predictions.shape.x != target.shape.x) {
			throw NNException("Predictions and target shapes don't match.");
		}

		Tensor3D dY;
		dY.shape = predictions.shape;
		dY.allocateCudaMemory();

		for (int i = 0; i < predictions.shape.x; i++) {
			dY.data[i] = - (predictions.data[i] - target.data[i]) / (static_cast<double>(1 - predictions.data[i]) * predictions.data[i]);
		}

		return dY;
	}

}
