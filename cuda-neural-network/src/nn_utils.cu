#include "nn_utils.hh"
#include "nn_exception.hh"

namespace nn_utils {

	void throwIfDeviceErrorsOccurred(const char* exception_message) {
		hipError_t error = hipGetLastError();
		if (error != hipSuccess) {
			throw NNException(exception_message);
		}
	}

}
